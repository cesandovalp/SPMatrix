#include "hip/hip_runtime.h"
#include "Kernels.hpp"

__global__
void TestKernel( double* x, double* y, double* result, unsigned size )
{
  int start     = blockIdx.x * blockDim.x + threadIdx.x;
  int increment = blockDim.x * gridDim.x;

  for( int index = start; index < size; index += increment ) 
  {
    result[index] = x[index] + y[index];
  }
}

//template void GPUAddition_( double* host_a, double* host_b, double* host_result, int rows, int columns );
//template void GPUAddition_( float*  host_a, float*  host_b, float*  host_result, int rows, int columns );
//template void GPUAddition_( int*    host_a, int*    host_b, int*    host_result, int rows, int columns );
void GPUAddition_( double* host_a, double* host_b, double* host_result, int rows, int columns )
{
  GPUAddition<double>( host_a, host_b, host_result, rows, columns );
}

void GPUMultiplication_( double* host_a, double* host_b, double* host_result, int a_rows, int a_columns, int b_columns )
{
  GPUAddition<double>( host_a, host_b, host_result, a_rows, a_columns, b_columns );
}
